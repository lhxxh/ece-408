#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

//#include "ensemble.h"
//#include "arg_bundle.h"
//#include "lenkf.h"

// #define BLOCK_SIZE 16
#define BLOCK_SIZE 32

__global__
void compute_P_kernel(double *e, double *C, double *P,
                      int L, int N) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    double e_elem = 0;
    // for (int l = 0; l < L; l ++) {
    //     e_elem += e[col * L + l] * e[row * L + l];
    // }

    //////////////////////////////////////////////////// loop unrolling
    int l;
    for (l = 0 ; l < L%2 ; l++)
      e_elem +=  e[col * L + l] * e[row * L + l];

    for(;l < L; l += 2)
    {
      e_elem +=  e[col * L + l] * e[row * L + l];
      e_elem +=  e[col * L + l + 1] * e[row * L + l + 1];
    }

    /////////////////////////////////////////////////////
    if (col < N && row < N) {
        P[row * N + col] = C[row * N + col] * e_elem;
    }
}

__global__
void compute_P_HT_kernel(double *P, double *H, double *P_HT,
                         int M, int N, int L) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < N && col < M) {
        double Value = 0;
        // for (int i = 0; i < N; i ++) {
        //     Value += P[row * N + i] * H[col * N + i];
        // }
    /////////////////////////////////////////////////////////////  loop unrolling
        int i;
        for(i = 0; i < N % 2; i++)
          Value += P[row * N + i] * H[col * N + i];
        for(;i < N; i += 2)
        {
          Value += P[row * N + i] * H[col * N + i];
          Value += P[row * N + i + 1] * H[col * N + i + 1];
        }
    ////////////////////////////////////////////////////////////
        P_HT[row * M + col] = Value / double(L - 1);
    }
}

void load_matrix(const char *fname,
                 double *buffer,
                 int dim1,
                 int dim2) {
    FILE *fid;
    int r;

    printf("Loading %s into CPU memory...\n", fname);
    fid = fopen(fname, "r");
    assert(fid);

    r = fseek(fid, 0L, SEEK_END);
    int matrix_size = ftell(fid) / sizeof(double);
    assert(matrix_size == (dim1 * dim2));
    r = fseek(fid, 0L, SEEK_SET);

    r = fread(buffer, sizeof(double), matrix_size, fid);
    assert(r == matrix_size);
    printf("Size of %s is: %d\n", fname, matrix_size);
    fclose(fid);
}

void save_matrix(const char *fname,
                 double *buffer,
                 int dim1,
                 int dim2) {
    FILE *fid;
    int r;
    int matrix_size = dim1 * dim2;

    printf("Saving %s into file...\n", fname);
    fid = fopen(fname, "w");
    assert(fid);

    r = fwrite(buffer, sizeof(double), matrix_size, fid);
    assert(r == matrix_size);
    printf("Size of %s is: %d\n", fname, matrix_size);
    fclose(fid);
}

void do_compute_P_HT(const char *P_HT_fname,
                     const char *e_fname,
                     const char *H_fname,
                     const char *C_fname,
                     const char *N_c,
                     const char *L_c,
                     const char *M_c) {
    int N = atoi(N_c);
    int L = atoi(L_c);
    int M = atoi(M_c);
    double *e = (double *) malloc(N * L * sizeof(double));
    double *H = (double *) malloc(M * N * sizeof(double));
    double *C = (double *) malloc(N * N * sizeof(double));
    double *P_HT = (double *) malloc(N * M * sizeof(double));

    clock_t start, finish;

    printf("0. Problem size: N=%d, L=%d, M=%d\n\n", N, L, M);

    printf("1. Load data into CPU memory.\n");
    load_matrix(e_fname, e, N, L);
    load_matrix(H_fname, H, M, N);
    load_matrix(C_fname, C, N, N);

    printf("2. Allocate GPU memory.\n");
    double *e_device;
    double *H_device;
    double *C_device;

    double *P_device;
    double *P_HT_device;

    hipMalloc((void **) &e_device, N * L * sizeof(double));
    hipMalloc((void **) &H_device, M * N * sizeof(double));
    hipMalloc((void **) &C_device, N * N * sizeof(double));

    hipMalloc((void **) &P_device, N * N * sizeof(double));
    hipMalloc((void **) &P_HT_device, N * M * sizeof(double));

    printf("3. Write data into GPU memory.\n");
    start = clock();
    hipMemcpy(e_device, e, N * L * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(H_device, H, M * N * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(C_device, C, N * N * sizeof(double), hipMemcpyHostToDevice);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("4. Call GPU cuda kernel.\n");
    start = clock();
    dim3 DimGrid;
    dim3 DimBlock;

    DimGrid = dim3(ceil(N / float(BLOCK_SIZE)), ceil(N / float(BLOCK_SIZE)), 1);
    DimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    compute_P_kernel<<<DimGrid, DimBlock>>>(e_device, C_device, P_device, L, N);
    hipDeviceSynchronize();

    DimGrid = dim3(ceil(M / float(BLOCK_SIZE)), ceil(N / float(BLOCK_SIZE)), 1);
    DimBlock = dim3(BLOCK_SIZE, BLOCK_SIZE, 1);
    compute_P_HT_kernel<<<DimGrid, DimBlock>>>(P_device, H_device, P_HT_device, M, N, L);
    hipDeviceSynchronize();
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("5. Read results from GPU memory.\n");
    start = clock();
    hipMemcpy(P_HT, P_HT_device, N * M * sizeof(double), hipMemcpyDeviceToHost);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    //for (int i = 0; i < 3; i ++) {
    //    for (int j = 0; j < M; j ++) {
    //        printf("%.2f, ", *(P_HT + i * M + j));
    //    }
    //    printf("\n");
    //}

    printf("6. Save results to file.\n");
    save_matrix(P_HT_fname, P_HT, N, M);

    printf("7. De-allocate CPU and GPU memory.\n");
    hipFree(e_device);
    hipFree(H_device);
    hipFree(C_device);
    hipFree(P_device);
    hipFree(P_HT_device);

    free(e);
    free(H);
    free(C);
    free(P_HT);
}

/*
void compute_P_HT(arg_bundle *ab, const sparse_rcs *H,
                  const int row_H, const char name_H,
                  const int n_rows);


void do_compute_P_HT(const char *P_HT_fname,
                     const char *e_fname,
                     const char *H_fname,
                     const char *C_fname) {
    arg_bundle *ab;
    sparse_rcs *H;
    sparse_rcs *P_HT_rcs;
    int rank, i;

    ab = malloc(sizeof(arg_bundle));
    assert(ab);

    fprintf(stderr, "loading e from %s\n", e_fname);
    ab->e = ensemble_import(e_fname);
    fprintf(stderr, "N=%d  L=%d\n", ab->e->N, ab->e->L);

    ab->config = malloc(sizeof(lenkf_config));
    assert(ab->config);
    ab->config->N = ab->e->N;

    ab->P_HT = NULL;

    fprintf(stderr, "\n");
    fprintf(stderr, "loading H from %s\n", H_fname);
    H = sparse_rcs_import(H_fname);
    fprintf(stderr, "m=%d  n=%d  N=%d\n", H->m, H->n, H->N);

    fprintf(stderr, "\n");
    fprintf(stderr, "loading C from %s\n", C_fname);
    ab->C = sb_toe_r_import(C_fname);
    ab->C_it = sb_toe_r_nz_it_create(ab->C);
    rank = ab->C->dim->rank;
    fprintf(stderr, "rank=%d\n", rank);
    fprintf(stderr, "n_phy=[ ");
    for (i = 0; i < rank; i++) {
        fprintf(stderr, "%d ", ab->C->dim->n_phy[i]);
    }
    fprintf(stderr, "]\n");
    fprintf(stderr, "n    =[ ");
    for (i = 0; i < rank; i++) {
        fprintf(stderr, "%d ", ab->C->dim->n[i]);
    }
    fprintf(stderr, "]\n");
    fprintf(stderr, "N_phy=[ ");
    for (i = 0; i < rank; i++) {
        fprintf(stderr, "%d ", ab->C->dim->N_phy[i]);
    }
    fprintf(stderr, "]\n");
    fprintf(stderr, "N    =[ ");
    for (i = 0; i < rank; i++) {
        fprintf(stderr, "%d ", ab->C->dim->N[i]);
    }
    fprintf(stderr, "]\n");

    compute_P_HT(ab, H, 0, 'H', H->m);

    P_HT_rcs = sparse_lil_2_rcs(ab->P_HT);
    sparse_rcs_export(P_HT_fname, P_HT_rcs);

    sparse_rcs_destroy(&H);
    sparse_rcs_destroy(&P_HT_rcs);
    ensemble_destroy(&ab->e);
    sb_toe_r_destroy(&ab->C);
    sb_toe_r_nz_it_destroy(&ab->C_it);
    sparse_lil_destroy(&ab->P_HT);
    free(ab->config);
    free(ab);
}
*/

int main(int argc, char **argv) {
    if (argc != 8) {
        fprintf(stderr, "Usage %s <P_HT_fname> <e_fname> <H_fname> <C_fname>\n", argv[0]);
        return 1;
    }

    do_compute_P_HT(argv[1], argv[2], argv[3], argv[4], argv[5], argv[6], argv[7]);

    return 0;
}
