#include <stdlib.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <time.h>

#define TILE_SIZE 32
//#define MAX_L 16

__global__
void C_e_H_optim_compute_P_HT_kernel(double *e, double *C, double *H,
                                 int *H_indices, int *H_indptr,
                                 double *P_HT, int M, int N, int L, int n) {
    __shared__ double P_tile[TILE_SIZE][TILE_SIZE];
    //__shared__ double H_tile[TILE_SIZE][TILE_SIZE];

    int tile_row = threadIdx.y;
    int tile_col = threadIdx.x;

    int P_HT_row = blockIdx.y * blockDim.y + tile_row;
    int P_HT_col = blockIdx.x * blockDim.x + tile_col;

    int H_indptr_start = H_indptr[P_HT_col];
    int H_indptr_end = H_indptr[P_HT_col + 1];

    double P_HT_value = 0;
    for (int n = 0; n < N; n += TILE_SIZE) {
        int P_row = P_HT_row;
        int P_col = n + tile_col;
        //int H_row = P_HT_col;
        //int H_col = n + tile_row;

        int e_row_idx = P_row * L; 
        int e_col_idx = P_col * L;

        if (P_row < N && P_col < N) {
            double e_eT_value = 0;
            for (int l = 0; l < L; l ++) {
                e_eT_value += e[e_row_idx + l] * e[e_col_idx + l];
            }

            int diff_n = P_col / n * n - P_row / n * n;
            int C_idx = diff_n * 2 + n - 1 + P_col % n - P_row % n;
            P_tile[tile_row][tile_col] = C[C_idx] * e_eT_value;
        } else {
            P_tile[tile_row][tile_col] = 0;
        }

        __syncthreads();

        //if (H_row < M && H_col < N) {
        //    H_tile[tile_row][tile_col] = H[H_row * N + H_col];
        //} else {
        //    H_tile[tile_row][tile_col] = 0;
        //}

        //__syncthreads();
        
        for (int t = H_indptr_start; t < H_indptr_end; t ++) {
            int H_tile_col = H_indices[t] - n;
            if (H_tile_col < TILE_SIZE) {
                P_HT_value += P_tile[tile_row][H_tile_col] * H[t];
                H_indptr_start += 1;
            } else {
                break;
            }
        }

        //for (int t = 0; t < TILE_SIZE; t ++) {
        //    P_HT_value += P_tile[tile_row][t] * H_tile[t][tile_col];
        //}
    }

    if (P_HT_row < N && P_HT_col < M) {
        P_HT[P_HT_row * M + P_HT_col] = P_HT_value / (L - 1);
    }
}

/*
__global__
void C_e_optim_compute_P_HT_kernel(double *e, double *C, double *H,
                                 double *P_HT, int M, int N, int L) {
    __shared__ double P_tile[TILE_SIZE][TILE_SIZE];
    __shared__ double H_tile[TILE_SIZE][TILE_SIZE];

    int tile_row = threadIdx.y;
    int tile_col = threadIdx.x;

    int P_HT_row = blockIdx.y * blockDim.y + tile_row;
    int P_HT_col = blockIdx.x * blockDim.x + tile_col;

    double P_HT_value = 0;
    for (int n = 0; n < N; n += TILE_SIZE) {
        int P_row = P_HT_row;
        int P_col = n + tile_col;
        int H_row = P_HT_col;
        int H_col = n + tile_row;

        int e_row_idx = P_row * L; 
        int e_col_idx = P_col * L;

        if (P_row < N && P_col < N) {
            double e_eT_value = 0;
            for (int l = 0; l < L; l ++) {
                e_eT_value += e[e_row_idx + l] * e[e_col_idx + l];
            }

            P_tile[tile_row][tile_col] = C[N - 1 + P_col - P_row] * e_eT_value;
        } else {
            P_tile[tile_row][tile_col] = 0;
        }

        __syncthreads();

        if (H_row < M && H_col < N) {
            H_tile[tile_row][tile_col] = H[H_row * N + H_col];
        } else {
            H_tile[tile_row][tile_col] = 0;
        }

        __syncthreads();

        for (int t = 0; t < TILE_SIZE; t ++) {
            P_HT_value += P_tile[tile_row][t] * H_tile[t][tile_col];
        }
    }

    if (P_HT_row < N && P_HT_col < M) {
        P_HT[P_HT_row * M + P_HT_col] = P_HT_value / (L - 1);
    }
}

__global__
void C_optim_compute_P_HT_kernel(double *e, double *C, double *H,
                                   double *P_HT, int M, int N, int L) {
    __shared__ double P_tile[TILE_SIZE][TILE_SIZE];
    __shared__ double H_tile[TILE_SIZE][TILE_SIZE];
    __shared__ double e_row_tile[TILE_SIZE][MAX_L];
    __shared__ double e_col_tile[TILE_SIZE][MAX_L];

    int tile_row = threadIdx.y;
    int tile_col = threadIdx.x;

    int P_HT_row = blockIdx.y * blockDim.y + tile_row;
    int P_HT_col = blockIdx.x * blockDim.x + tile_col;

    double P_HT_value = 0;
    for (int n = 0; n < N; n += TILE_SIZE) {
        int P_row = P_HT_row;
        int P_col = n + tile_col;
        int H_row = P_HT_col;
        int H_col = n + tile_row;

        int e_row_idx = P_row * L; 
        for (int l = 0; l < L; l += TILE_SIZE) {
            e_row_tile[tile_row][l + tile_col] = e[e_row_idx + l + tile_col];
        }

        int e_col_idx = P_col * L;
        for (int l = 0; l < L; l += TILE_SIZE) {
            e_col_tile[tile_col][l + tile_row] = e[e_col_idx + l + tile_row];
        }

        __syncthreads();

        if (P_row < N && P_col < N) {
            double e_eT_value = 0;
            for (int l = 0; l < L; l ++) {
                e_eT_value += e_row_tile[tile_row][l] * e_col_tile[tile_col][l];
            }

            P_tile[tile_row][tile_col] = C[N - 1 + P_col - P_row] * e_eT_value;
        } else {
            P_tile[tile_row][tile_col] = 0;
        }

        if (H_row < M && H_col < N) {
            H_tile[tile_row][tile_col] = H[H_row * N + H_col];
        } else {
            H_tile[tile_row][tile_col] = 0;
        }

        __syncthreads();

        for (int t = 0; t < TILE_SIZE; t ++) {
            P_HT_value += P_tile[tile_row][t] * H_tile[t][tile_col];
        }
    }

    if (P_HT_row < N && P_HT_col < M) {
        P_HT[P_HT_row * M + P_HT_col] = P_HT_value / (L - 1);
    }
}

__global__
void compute_P_kernel(double *e, double *C, double *P,
                      int L, int N) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;
    
    double e_elem = 0;
    for (int l = 0; l < L; l ++) {
        e_elem += e[col * L + l] * e[row * L + l];
    }

    if (col < N && row < N) {
        P[row * N + col] = C[row * N + col] * e_elem;
    }
}

__global__
void compute_P_HT_kernel(double *P, double *H, double *P_HT,
                         int M, int N, int L) {

    int col = threadIdx.x + blockIdx.x * blockDim.x;
    int row = threadIdx.y + blockIdx.y * blockDim.y;

    if (row < N && col < M) {
        double Value = 0;
        for (int i = 0; i < N; i ++) {
            Value += P[row * N + i] * H[col * N + i];
        }
        P_HT[row * M + col] = Value / double(L - 1);
    }
}
*/

int load_sparse_matrix_nnz(const char *fname,
                           int dim1,
                           int dim2) {
    FILE *fid;
    int sizeof_elem;
    int m, n, nnz;
    int r;

    printf("Loading %s into CPU memory...\n", fname);
    fid = fopen(fname, "r");
    assert(fid);

    r = fread(&sizeof_elem, sizeof(int), 1, fid);
    assert(sizeof_elem == sizeof(double));
    r = fread(&m, sizeof(int), 1, fid);
    assert(m == dim1);
    r = fread(&n, sizeof(int), 1, fid);
    assert(n == dim2);
    r = fread(&nnz, sizeof(int), 1, fid);
    assert(r == 1);
    fclose(fid);
    
    return nnz;
}

void load_sparse_matrix(const char *fname,
                        double *buffer,
                        int *indices,
                        int *indptr,
                        int dim1,
                        int dim2) {
    FILE *fid;
    int sizeof_elem;
    int m, n, nnz;
    int r;

    printf("Loading %s into CPU memory...\n", fname);
    fid = fopen(fname, "r");
    assert(fid);

    r = fread(&sizeof_elem, sizeof(int), 1, fid);
    assert(sizeof_elem == sizeof(double));
    r = fread(&m, sizeof(int), 1, fid);
    assert(m == dim1);
    r = fread(&n, sizeof(int), 1, fid);
    assert(n == dim2);
    r = fread(&nnz, sizeof(int), 1, fid);
    assert(r == 1);

    r = fread(buffer, sizeof(double), nnz, fid);
    assert(r == nnz);
    r = fread(indices, sizeof(int), nnz, fid);
    assert(r == nnz);
    r = fread(indptr, sizeof(int), m + 1, fid);
    assert(r == m + 1);

    printf("Size of %s is: %d\n", fname, nnz);
    fclose(fid);
}

void load_matrix(const char *fname,
                 double *buffer,
                 int dim1,
                 int dim2) {
    FILE *fid;
    int r;

    printf("Loading %s into CPU memory...\n", fname);
    fid = fopen(fname, "r");
    assert(fid);
    
    r = fseek(fid, 0L, SEEK_END);
    int matrix_size = ftell(fid) / sizeof(double);
    assert(matrix_size == (dim1 * dim2));
    r = fseek(fid, 0L, SEEK_SET);

    r = fread(buffer, sizeof(double), matrix_size, fid);
    assert(r == matrix_size);
    printf("Size of %s is: %d\n", fname, matrix_size);
    fclose(fid);
}

void save_matrix(const char *fname,
                 double *buffer,
                 int dim1,
                 int dim2) {
    FILE *fid;
    int r;
    int matrix_size = dim1 * dim2;

    printf("Saving %s into file...\n", fname);
    fid = fopen(fname, "w");
    assert(fid);

    r = fwrite(buffer, sizeof(double), matrix_size, fid);
    assert(r == matrix_size);
    printf("Size of %s is: %d\n", fname, matrix_size);
    fclose(fid);
}

/*
void do_compute_P_HT(const char *P_HT_fname,
                     const char *e_fname,
                     const char *H_fname,
                     const char *C_fname,
                     const char *N_c,
                     const char *L_c,
                     const char *M_c) {
    int N = atoi(N_c);
    int L = atoi(L_c);
    int M = atoi(M_c);
    double *e = (double *) malloc(N * L * sizeof(double));
    double *H = (double *) malloc(M * N * sizeof(double));
    double *C = (double *) malloc(N * N * sizeof(double));
    double *P_HT = (double *) malloc(N * M * sizeof(double));

    clock_t start, finish;

    printf("0. Problem size: N=%d, L=%d, M=%d\n\n", N, L, M);

    printf("1. Load data into CPU memory.\n");
    load_matrix(e_fname, e, N, L);
    load_matrix(H_fname, H, M, N);
    load_matrix(C_fname, C, N, N);

    printf("2. Allocate GPU memory.\n");
    double *e_device;
    double *H_device;
    double *C_device;

    double *P_device;
    double *P_HT_device;

    cudaMalloc((void **) &e_device, N * L * sizeof(double));
    cudaMalloc((void **) &H_device, M * N * sizeof(double));
    cudaMalloc((void **) &C_device, N * N * sizeof(double));

    cudaMalloc((void **) &P_device, N * N * sizeof(double));
    cudaMalloc((void **) &P_HT_device, N * M * sizeof(double));

    printf("3. Write data into GPU memory.\n");
    start = clock();
    cudaMemcpy(e_device, e, N * L * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(H_device, H, M * N * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(C_device, C, N * N * sizeof(double), cudaMemcpyHostToDevice);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("4. Call GPU cuda kernel.\n");
    start = clock();
    dim3 DimGrid;
    dim3 DimBlock;

    DimGrid = dim3(ceil(N / float(TILE_SIZE)), ceil(N / float(TILE_SIZE)), 1);
    DimBlock = dim3(TILE_SIZE, TILE_SIZE, 1);
    compute_P_kernel<<<DimGrid, DimBlock>>>(e_device, C_device, P_device, L, N);
    cudaDeviceSynchronize();

    DimGrid = dim3(ceil(M / float(TILE_SIZE)), ceil(N / float(TILE_SIZE)), 1);
    DimBlock = dim3(TILE_SIZE, TILE_SIZE, 1);
    compute_P_HT_kernel<<<DimGrid, DimBlock>>>(P_device, H_device, P_HT_device, M, N, L);
    cudaDeviceSynchronize();
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("5. Read results from GPU memory.\n");
    start = clock();
    cudaMemcpy(P_HT, P_HT_device, N * M * sizeof(double), cudaMemcpyDeviceToHost);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("6. Save results to file.\n");
    save_matrix(P_HT_fname, P_HT, N, M);

    printf("7. De-allocate CPU and GPU memory.\n");
    cudaFree(e_device);
    cudaFree(H_device);
    cudaFree(C_device);
    cudaFree(P_device);
    cudaFree(P_HT_device);

    free(e);
    free(H);
    free(C);
    free(P_HT);
}

void do_optim_compute_P_HT(const char *P_HT_fname,
                     const char *e_fname,
                     const char *H_fname,
                     const char *C_fname,
                     const char *N_c,
                     const char *L_c,
                     const char *M_c) {
    int N = atoi(N_c);
    int L = atoi(L_c);
    int M = atoi(M_c);
    double *e = (double *) malloc(N * L * sizeof(double));
    double *H = (double *) malloc(M * N * sizeof(double));
    double *C = (double *) malloc(2 * N * sizeof(double));
    double *P_HT = (double *) malloc(N * M * sizeof(double));

    clock_t start, finish;

    printf("0. Problem size: N=%d, L=%d, M=%d\n\n", N, L, M);

    printf("1. Load data into CPU memory.\n");
    load_matrix(e_fname, e, N, L);
    load_matrix(H_fname, H, M, N);
    load_matrix(C_fname, C, 2, N);

    printf("2. Allocate GPU memory.\n");
    double *e_device;
    double *H_device;
    double *C_device;
    double *P_HT_device;

    cudaMalloc((void **) &e_device, N * L * sizeof(double));
    cudaMalloc((void **) &H_device, M * N * sizeof(double));
    cudaMalloc((void **) &C_device, 2 * N * sizeof(double));
    cudaMalloc((void **) &P_HT_device, N * M * sizeof(double));

    printf("3. Write data into GPU memory.\n");
    start = clock();
    cudaMemcpy(e_device, e, N * L * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(H_device, H, M * N * sizeof(double), cudaMemcpyHostToDevice);
    cudaMemcpy(C_device, C, 2 * N * sizeof(double), cudaMemcpyHostToDevice);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("4. Call GPU cuda kernel.\n");
    start = clock();
    dim3 DimGrid;
    dim3 DimBlock;

    DimGrid = dim3(ceil(N / float(TILE_SIZE)), ceil(N / float(TILE_SIZE)), 1);
    DimBlock = dim3(TILE_SIZE, TILE_SIZE, 1);
    C_e_optim_compute_P_HT_kernel<<<DimGrid, DimBlock>>>(e_device, C_device, H_device, P_HT_device, M, N, L);
    cudaDeviceSynchronize();
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("5. Read results from GPU memory.\n");
    start = clock();
    cudaMemcpy(P_HT, P_HT_device, N * M * sizeof(double), cudaMemcpyDeviceToHost);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("6. Save results to file.\n");
    save_matrix(P_HT_fname, P_HT, N, M);

    printf("7. De-allocate CPU and GPU memory.\n");
    cudaFree(e_device);
    cudaFree(H_device);
    cudaFree(C_device);
    cudaFree(P_HT_device);

    free(e);
    free(H);
    free(C);
    free(P_HT);
}
*/

void do_C_e_H_optim_compute_P_HT(const char *P_HT_fname,
                     const char *e_fname,
                     const char *H_fname,
                     const char *C_fname,
                     const char *N_c,
                     const char *L_c,
                     const char *M_c,
                     const char *n_c) {
    int N = atoi(N_c);
    int L = atoi(L_c);
    int M = atoi(M_c);
    int n = atoi(n_c);
    double *e = (double *) malloc(N * L * sizeof(double));
    double *C = (double *) malloc(2 * N * sizeof(double));
    double *P_HT = (double *) malloc(N * M * sizeof(double));

    clock_t start, finish;

    printf("0. Problem size: N=%d, L=%d, M=%d\n\n", N, L, M);

    printf("1. Load data into CPU memory.\n");
    int H_nnz = load_sparse_matrix_nnz(H_fname, M, N);
    double *H = (double *) malloc(H_nnz * sizeof(double));
    int *H_indices = (int *) malloc(H_nnz * sizeof(int));
    int *H_indptr = (int *) malloc((M + 1) * sizeof(int));

    load_matrix(e_fname, e, N, L);
    load_sparse_matrix(H_fname, H, H_indices, H_indptr, M, N);
    load_matrix(C_fname, C, 2, N);

    printf("2. Allocate GPU memory.\n");
    double *e_device;
    double *H_device;
    int *H_indices_device;
    int *H_indptr_device;
    double *C_device;
    double *P_HT_device;

    hipMalloc((void **) &e_device, N * L * sizeof(double));
    hipMalloc((void **) &H_device, H_nnz * sizeof(double));
    hipMalloc((void **) &H_indices_device, H_nnz * sizeof(int));
    hipMalloc((void **) &H_indptr_device, (M + 1) * sizeof(int));
    hipMalloc((void **) &C_device, 2 * N * sizeof(double));
    hipMalloc((void **) &P_HT_device, N * M * sizeof(double));

    printf("3. Write data into GPU memory.\n");
    start = clock();
    hipMemcpy(e_device, e, N * L * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(H_device, H, H_nnz * sizeof(double), hipMemcpyHostToDevice);
    hipMemcpy(H_indices_device, H_indices, H_nnz * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(H_indptr_device, H_indptr, (M + 1) * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(C_device, C, 2 * N * sizeof(double), hipMemcpyHostToDevice);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("4. Call GPU cuda kernel.\n");
    start = clock();
    dim3 DimGrid;
    dim3 DimBlock;

    DimGrid = dim3(ceil(M / float(TILE_SIZE)), ceil(N / float(TILE_SIZE)), 1);
    DimBlock = dim3(TILE_SIZE, TILE_SIZE, 1);
    C_e_H_optim_compute_P_HT_kernel<<<DimGrid, DimBlock>>>(e_device, C_device, H_device, H_indices_device, H_indptr_device, P_HT_device, M, N, L, n);
    hipDeviceSynchronize();
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("5. Read results from GPU memory.\n");
    start = clock();
    hipMemcpy(P_HT, P_HT_device, N * M * sizeof(double), hipMemcpyDeviceToHost);
    finish = clock();
    printf("Latency: %fms\n", (double)(finish - start) * 1000 / CLOCKS_PER_SEC);

    printf("6. Save results to file.\n");
    save_matrix(P_HT_fname, P_HT, N, M);

    printf("7. De-allocate CPU and GPU memory.\n");
    hipFree(e_device);
    hipFree(H_device);
    hipFree(H_indices_device);
    hipFree(H_indptr_device);
    hipFree(C_device);
    hipFree(P_HT_device);

    free(e);
    free(H);
    free(H_indices);
    free(H_indptr);
    free(C);
    free(P_HT);
}

int main(int argc, char **argv) {
    //if (argc != 8) {
    //    fprintf(stderr, "Usage %s <P_HT_fname> <e_fname> <H_fname> <C_fname>\n", argv[0]);
    //    return 1;
    //}

    //do_compute_P_HT(argv[1], argv[2], argv[3], argv[4], argv[5], argv[6], argv[7]);
    //do_optim_compute_P_HT(argv[1], argv[2], argv[3], argv[4], argv[5], argv[6], argv[7]);
    do_C_e_H_optim_compute_P_HT(argv[1], argv[2], argv[3], argv[4], argv[5], argv[6], argv[7], argv[8]);
    return 0;
}
